#include "hip/hip_runtime.h"
//
//  Copyright (c) 2021, NVIDIA Corporation.  All rights reserved.
//
//  NVIDIA Corporation and its licensors retain all intellectual property and proprietary
//  rights in and to this software, related documentation and any modifications thereto.
//  Any use, reproduction, disclosure or distribution of this software and related
//  documentation without an express license agreement from NVIDIA Corporation is strictly
//  prohibited.
//
//  TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
//  AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
//  INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
//  PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
//  SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
//  LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
//  BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
//  INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
//  SUCH DAMAGES
//

#include "DemandLoaderTestKernels.h"

#include "Util/Exception.h"

#include <DemandLoading/Texture2D.h>

using namespace demandLoading;

__global__ static void pageRequester( DeviceContext context, unsigned int pageId, bool* isResident )
{
    pagingMapOrRequest( context, pageId, isResident );
}

__host__ void launchPageRequester( hipStream_t stream, const DeviceContext& context, unsigned int pageId, bool* isResident )
{
    pageRequester<<<1, 1, 0U, stream>>>( context, pageId, isResident );
    DEMAND_CUDA_CHECK( hipGetLastError() );
}
